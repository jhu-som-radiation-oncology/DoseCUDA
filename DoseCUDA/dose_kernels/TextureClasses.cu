#include "TextureClasses.cuh"
#include "CudaClasses.cuh"


void Texture3D::makeArray(const PointIJK &size)
{
    auto chdesc = hipCreateChannelDesc<float>();
    auto extent = make_hipExtent(size.i, size.j, size.k);

    CUDA_CHECK(hipMalloc3DArray(&m_arr, &chdesc, extent, hipArrayDefault));
}


void Texture3D::makeTexture(hipTextureFilterMode filterMode, float border)
{
    hipResourceDesc rsdesc{ };
    hipTextureDesc txdesc{ };

    rsdesc.resType = hipResourceTypeArray;
    rsdesc.res.array.array = m_arr;

    txdesc.addressMode[0]               = hipAddressModeBorder;
    txdesc.addressMode[1]               = hipAddressModeBorder;
    txdesc.addressMode[2]               = hipAddressModeBorder;
    txdesc.filterMode                   = filterMode;
    txdesc.readMode                     = hipReadModeElementType;
    txdesc.sRGB                         = false;
    txdesc.borderColor[0]               = border;
    txdesc.borderColor[1]               = 0.0f;
    txdesc.borderColor[2]               = 0.0f;
    txdesc.borderColor[3]               = 0.0f;
    txdesc.normalizedCoords             = false;
    txdesc.maxAnisotropy                = 0;
    txdesc.mipmapFilterMode             = hipFilterModePoint;
    txdesc.mipmapLevelBias              = 0.0f;
    txdesc.minMipmapLevelClamp          = 0.0f;
    txdesc.maxMipmapLevelClamp          = 0.0f;
    txdesc.disableTrilinearOptimization = false;

    CUDA_CHECK(hipCreateTextureObject(&m_tex, &rsdesc, &txdesc, nullptr));
}


Texture3D::Texture3D(const float data[], const PointIJK &size, hipTextureFilterMode filterMode, float border, hipMemcpyKind direction):
    m_arr(NULL),
    m_tex(0),
    m_isCopy(false)
{
    this->makeArray(size);

    hipMemcpy3DParms params{ };
    params.srcPtr   = make_hipPitchedPtr((void *)data, size.i * sizeof *data, size.i, size.j);
    params.dstArray = m_arr;
    params.extent   = make_hipExtent(size.i, size.j, size.k);
    params.kind     = direction;
    CUDA_CHECK(hipMemcpy3D(&params));

    this->makeTexture(filterMode, border);
}


Texture3D::~Texture3D()
{
    if (!m_isCopy) {
        hipDestroyTextureObject(m_tex);
        hipFreeArray(m_arr);
    }
}
