#include "TextureClasses.cuh"
#include "CudaClasses.cuh"


void Texture3D::makeArray(const PointIJK &size)
{
    auto chdesc = hipCreateChannelDesc<float>();
    auto extent = make_hipExtent(size.i, size.j, size.k);

    CUDA_CHECK(hipMalloc3DArray(&m_arr, &chdesc, extent, hipArrayDefault));
}


void Texture3D::makeTexture(hipTextureFilterMode filterMode)
{
    hipResourceDesc rsdesc{ };
    hipTextureDesc txdesc{ };

    rsdesc.resType = hipResourceTypeArray;
    rsdesc.res.array.array = m_arr;

    txdesc.addressMode[0]               = hipAddressModeClamp;
    txdesc.addressMode[1]               = hipAddressModeClamp;
    txdesc.addressMode[2]               = hipAddressModeClamp;
    txdesc.filterMode                   = filterMode;
    txdesc.readMode                     = hipReadModeElementType;
    txdesc.sRGB                         = false;
    txdesc.borderColor[0]               = 0.0f;
    txdesc.borderColor[1]               = 0.0f;
    txdesc.borderColor[2]               = 0.0f;
    txdesc.borderColor[3]               = 0.0f;
    txdesc.normalizedCoords             = false;
    txdesc.maxAnisotropy                = 0;
    txdesc.mipmapFilterMode             = hipFilterModePoint;
    txdesc.mipmapLevelBias              = 0.0f;
    txdesc.minMipmapLevelClamp          = 0.0f;
    txdesc.maxMipmapLevelClamp          = 0.0f;
    txdesc.disableTrilinearOptimization = false;

    CUDA_CHECK(hipCreateTextureObject(&m_tex, &rsdesc, &txdesc, nullptr));
}


Texture3D::Texture3D(const float data[], const PointIJK &size, hipTextureFilterMode filterMode, hipMemcpyKind direction):
    m_arr(NULL),
    m_tex(0),
    m_isCopy(false)
{
    this->makeArray(size);

    hipMemcpy3DParms params{ };
    params.srcPtr   = make_hipPitchedPtr((void *)data, size.i * sizeof *data, size.i, size.j);
    params.dstArray = m_arr;
    params.extent   = make_hipExtent(size.i, size.j, size.k);
    params.kind     = direction;
    CUDA_CHECK(hipMemcpy3D(&params));

    this->makeTexture(filterMode);
}


Texture3D::~Texture3D()
{
    if (!m_isCopy) {
        hipDestroyTextureObject(m_tex);
        hipFreeArray(m_arr);
    }
}
