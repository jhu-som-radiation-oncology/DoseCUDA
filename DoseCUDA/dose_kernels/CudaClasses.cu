#include "hip/hip_runtime.h"
#include "./CudaClasses.cuh"


__host__ CudaBeam::CudaBeam(BeamClass * h_beam) : BeamClass(h_beam) {}

__device__ void CudaBeam::unitVectorToSource(const PointXYZ * point_xyz, PointXYZ * uvec) {

	float dx = this->src.x - point_xyz->x;
	float dy = this->src.y - point_xyz->y;
	float dz = this->src.z - point_xyz->z;

	float r = rnorm3df(dx, dy, dz);

	uvec->x = dx * r;
	uvec->y = dy * r;
	uvec->z = dz * r;
}

__device__ float CudaBeam::distanceToSource(const PointXYZ * point_xyz){
	
	float dx = this->src.x - point_xyz->x;
	float dy = this->src.y - point_xyz->y;
	float dz = this->src.z - point_xyz->z;

	float r = norm3df(dx, dy, dz);

	return r;

}

__host__ CudaDose::CudaDose(DoseClass * h_dose) : DoseClass(h_dose) {}

__device__ void CudaDose::pointIJKtoXYZ(const PointIJK * point_ijk, PointXYZ * point_xyz, BeamClass * beam) {

	point_xyz->x = (float)point_ijk->i * this->spacing - beam->iso.x;
	point_xyz->y = (float)point_ijk->j * this->spacing - beam->iso.y;
	point_xyz->z = (float)point_ijk->k * this->spacing - beam->iso.z;

}

/** sincos but with a value theoretically supplied to arctangent */
__device__ void sincos_from_atan(float y, float x, float *sptr, float *cptr) {

	float slope = y / x;

	*cptr = rsqrtf(fmaf(slope, slope, 1.0));
	*sptr = slope * *cptr;
}

__device__ void CudaDose::pointXYZImageToHead(const PointXYZ * point_img, PointXYZ * point_head, BeamClass * beam){

	float sinx, cosx;

	//table rotation - rotate about y-axis
	float xt, yt, zt;
	sinx = -beam->sinta;
	cosx = beam->costa;
	xt = point_img->x * cosx + point_img->z * sinx;
	yt = point_img->y;
	zt = -point_img->x * sinx + point_img->z * cosx;

	//gantry rotation - rotate about z-axis
	float xg, yg, zg;
	sinx = -beam->singa;
	cosx = beam->cosga;
	xg = xt * cosx - yt * sinx;
	yg = xt * sinx + yt * cosx;
	zg = zt;

	//collimator rotation = rotate about y-axis
	float xc, yc, zc;
	sinx = -beam->sinca;
	cosx = beam->cosca;
	xc  = xg * cosx + zg * sinx;
	yc = yg;
	zc = -xg * sinx + zg * cosx;


	//swap final coordinates to match DICOM nozzle coordinate system
	//for an AP beam:
	//	beam travels in negative z direction
	//	positive x is to the patient's left
	//	positive y is to the patient's superior
	point_head->x = -xc;
	point_head->y = zc;
	point_head->z = yc;

}

__device__ void CudaDose::pointXYZHeadToImage(const PointXYZ * point_head, PointXYZ * point_img, BeamClass * beam){

	float sinx, cosx;

	//convert back to DICOM patient LPS coordinates
	float xz = -point_head->x;
	float yz = point_head->z;
	float zz = point_head->y;

	//collimator rotation = rotate about y-axis (again, negative direction)
	float xc, yc, zc;
	sinx = beam->sinca;
	cosx = beam->cosca;
	xc  = xz * cosx + zz * sinx;
	yc = yz;
	zc = -xz * sinx + zz * cosx;

	//gantry rotation - rotate about z-axis (negative direction)
	float xg, yg, zg;
	sinx = beam->singa;
	cosx = beam->cosga;
	xg = xc * cosx - yc * sinx;
	yg = xc * sinx + yc * cosx;
	zg = zc;

	//table rotation - rotate about y-axis (negative direction)
	float xt, yt, zt;
	sinx = beam->sinta;
	cosx = beam->costa;
	xt = xg * cosx + zg * sinx;
	yt = yg;
	zt = -xg * sinx + zg * cosx;

	point_img->x = xt;
	point_img->y = yt;
	point_img->z = zt;

}

__device__ void CudaDose::pointXYZClosestCAXPoint(const PointXYZ * point_xyz, PointXYZ * point_cax, BeamClass * beam){

	float d1x = beam->iso.x - beam->src.x;
	float d1y = beam->iso.y - beam->src.y;
	float d1z = beam->iso.z - beam->src.z;

	float d2x = point_xyz->x - beam->src.x;
	float d2y = point_xyz->y - beam->src.y;
	float d2z = point_xyz->z - beam->src.z;

	float t = (d1x * d2x + d1y * d2y + d1z * d2z) / (d1x * d1x + d1y * d1y + d1z * d1z);

	point_cax->x = fmaf(t, d1x, beam->src.x);
	point_cax->y = fmaf(t, d1y, beam->src.y);
	point_cax->z = fmaf(t, d1z, beam->src.z);

}

__device__ float CudaDose::pointXYZDistanceToCAX(const PointXYZ * point_head_xyz){

	return hypotf(point_head_xyz->x, point_head_xyz->z);

}

__device__ float CudaDose::pointXYZDistanceToSource(const PointXYZ * point_img_xyz, BeamClass * beam){

	float dx = beam->src.x - point_img_xyz->x;
	float dy = beam->src.y - point_img_xyz->y;
	float dz = beam->src.z - point_img_xyz->z;

	return norm3df(dx, dy, dz);

}

__global__ void rayTraceKernel(CudaDose * dose, CudaBeam * beam, Texture3D DensityTexture){

	PointIJK vox_ijk;
	vox_ijk.k = threadIdx.x + (blockIdx.x * blockDim.x);
	vox_ijk.j = threadIdx.y + (blockIdx.y * blockDim.y);
	vox_ijk.i = threadIdx.z + (blockIdx.z * blockDim.z);

	if(!dose->pointIJKWithinImage(&vox_ijk)) {
		return;
	}

	int vox_index = dose->pointIJKtoIndex(&vox_ijk);

	PointXYZ vox_xyz;
	dose->pointIJKtoXYZ(&vox_ijk, &vox_xyz, beam);

	PointXYZ uvec;
	beam->unitVectorToSource(&vox_xyz, &uvec);

	PointXYZ vox_ray_xyz, tex_xyz;
	PointIJK vox_ray_ijk;

	int vox_ray_index = 0;
    float ray_length = 0.0;
    float wet_sum = -0.05;
    float density = 0.0;
	const float step_length = 1.0;

    while(true){

		vox_ray_xyz.x = fmaf(uvec.x, ray_length, vox_xyz.x);
		vox_ray_xyz.y = fmaf(uvec.y, ray_length, vox_xyz.y);
		vox_ray_xyz.z = fmaf(uvec.z, ray_length, vox_xyz.z);

		dose->pointXYZtoTextureXYZ(&vox_ray_xyz, &tex_xyz, beam);
		if (!dose->textureXYZWithinImage(&tex_xyz)) {
			break;
		}
		density = DensityTexture.sample(tex_xyz);

		wet_sum = fmaf(fmaxf(density, 0.0), step_length / 10.0, wet_sum);

		ray_length += step_length;

	}

	dose->WETArray[vox_index] = wet_sum;

    __syncthreads();

}
