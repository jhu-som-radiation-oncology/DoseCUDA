#include "hip/hip_runtime.h"
#include "./CudaClasses.cuh"


__host__ CudaBeam::CudaBeam(BeamClass * h_beam) : BeamClass(h_beam) {}

__device__ void CudaBeam::unitVectorToSource(const PointXYZ * point_xyz, PointXYZ * uvec) {

	float dx = this->src.x - point_xyz->x;
	float dy = this->src.y - point_xyz->y;
	float dz = this->src.z - point_xyz->z;

	float r = rnorm3df(dx, dy, dz);

	uvec->x = dx * r;
	uvec->y = dy * r;
	uvec->z = dz * r;
}

__device__ float CudaBeam::distanceToSource(const PointXYZ * point_xyz){
	
	float dx = this->src.x - point_xyz->x;
	float dy = this->src.y - point_xyz->y;
	float dz = this->src.z - point_xyz->z;

	float r = norm3df(dx, dy, dz);

	return r;

}

__host__ CudaDose::CudaDose(DoseClass * h_dose) : DoseClass(h_dose) {}

__device__ bool CudaDose::pointIJKWithinImage(const PointIJK * point_ijk) {

	return point_ijk->i < this->img_sz.i
	    && point_ijk->j < this->img_sz.j
		&& point_ijk->k < this->img_sz.k;
}

__device__ unsigned int CudaDose::pointIJKtoIndex(const PointIJK * point_ijk) {

	return point_ijk->i + this->img_sz.i * (point_ijk->j + this->img_sz.j * point_ijk->k);
}

__device__ void CudaDose::pointIJKtoXYZ(const PointIJK * point_ijk, PointXYZ * point_xyz, BeamClass * beam) {

	point_xyz->x = (float)point_ijk->i * this->spacing - beam->iso.x;
	point_xyz->y = (float)point_ijk->j * this->spacing - beam->iso.y;
	point_xyz->z = (float)point_ijk->k * this->spacing - beam->iso.z;

}

__device__ void CudaDose::pointXYZtoIJK(const PointXYZ * point_xyz, PointIJK * point_ijk, BeamClass * beam) {

	point_ijk->i = (int)roundf((point_xyz->x + beam->iso.x) / this->spacing);
	point_ijk->j = (int)roundf((point_xyz->y + beam->iso.y) / this->spacing);
	point_ijk->k = (int)roundf((point_xyz->z + beam->iso.z) / this->spacing);
}

/** sincos but with a value theoretically supplied to arctangent */
__device__ void sincos_from_atan(float y, float x, float *sptr, float *cptr) {

	float slope = y / x;

	*cptr = rsqrtf(fmaf(slope, slope, 1.0));
	*sptr = slope * *cptr;
}

__device__ void CudaDose::pointXYZImageToHead(const PointXYZ * point_img, PointXYZ * point_head, BeamClass * beam){

	float sinx, cosx;

	//table rotation - rotate about y-axis
	float xt, yt, zt;
	sinx = -beam->sinta;
	cosx = beam->costa;
	xt = point_img->x * cosx + point_img->z * sinx;
	yt = point_img->y;
	zt = -point_img->x * sinx + point_img->z * cosx;

	//gantry rotation - rotate about z-axis
	float xg, yg, zg;
	sinx = -beam->singa;
	cosx = beam->cosga;
	xg = xt * cosx - yt * sinx;
	yg = xt * sinx + yt * cosx;
	zg = zt;

	//collimator rotation = rotate about y-axis
	float xc, yc, zc;
	sinx = -beam->sinca;
	cosx = beam->cosca;
	xc  = xg * cosx + zg * sinx;
	yc = yg;
	zc = -xg * sinx + zg * cosx;

	point_head->x = xc;
	point_head->y = yc;
	point_head->z = zc;

}

__device__ void CudaDose::pointXYZHeadToImage(const PointXYZ * point_head, PointXYZ * point_img, BeamClass * beam){

	float sinx, cosx;

	float xz = point_head->x;
	float yz = point_head->y;
	float zz = point_head->z;

	//collimator rotation = rotate about y-axis (again, negative direction)
	float xc, yc, zc;
	sinx = beam->sinca;
	cosx = beam->cosca;
	xc  = xz * cosx + zz * sinx;
	yc = yz;
	zc = -xz * sinx + zz * cosx;

	//gantry rotation - rotate about z-axis (negative direction)
	float xg, yg, zg;
	sinx = beam->singa;
	cosx = beam->cosga;
	xg = xc * cosx - yc * sinx;
	yg = xc * sinx + yc * cosx;
	zg = zc;

	//table rotation - rotate about y-axis (negative direction)
	float xt, yt, zt;
	sinx = beam->sinta;
	cosx = beam->costa;
	xt = xg * cosx + zg * sinx;
	yt = yg;
	zt = -xg * sinx + zg * cosx;

	point_img->x = xt;
	point_img->y = yt;
	point_img->z = zt;

}

__device__ void CudaDose::pointXYZClosestCAXPoint(const PointXYZ * point_xyz, PointXYZ * point_cax, BeamClass * beam){

	float d1x = beam->iso.x - beam->src.x;
	float d1y = beam->iso.y - beam->src.y;
	float d1z = beam->iso.z - beam->src.z;

	float d2x = point_xyz->x - beam->src.x;
	float d2y = point_xyz->y - beam->src.y;
	float d2z = point_xyz->z - beam->src.z;

	float t = (d1x * d2x + d1y * d2y + d1z * d2z) / (d1x * d1x + d1y * d1y + d1z * d1z);

	point_cax->x = fmaf(t, d1x, beam->src.x);
	point_cax->y = fmaf(t, d1y, beam->src.y);
	point_cax->z = fmaf(t, d1z, beam->src.z);

}

__device__ float CudaDose::pointXYZDistanceToCAX(const PointXYZ * point_head_xyz){

	return hypotf(point_head_xyz->x, point_head_xyz->z);

}

__device__ float CudaDose::pointXYZDistanceToSource(const PointXYZ * point_img_xyz, BeamClass * beam){

	float dx = beam->src.x - point_img_xyz->x;
	float dy = beam->src.y - point_img_xyz->y;
	float dz = beam->src.z - point_img_xyz->z;

	return norm3df(dx, dy, dz);

}