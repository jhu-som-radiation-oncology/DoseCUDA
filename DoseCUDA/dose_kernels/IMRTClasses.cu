#include "hip/hip_runtime.h"
#include "./IMRTClasses.cuh"
#ifndef DOSECUDA_DEVICE_POINTER
#	define DOSECUDA_DEVICE_POINTER
#endif
#include "MemoryClasses.h"


__host__ IMRTBeam::IMRTBeam(BeamClass * h_beam) : CudaBeam(h_beam) {}

__device__ float IMRTBeam::headTransmission(const PointXYZ* point_xyz, const float iso_to_source, const float xSigma, const float ySigma){

	const float mlc_scale = (PRIMARY_SOURCE_DISTANCE - MLC_DISTANCE) / PRIMARY_SOURCE_DISTANCE;
	const float divergence_scale = (iso_to_source - point_xyz->z) / (MLC_DISTANCE - point_xyz->z);
	const float invSqrt2_x = 1.0f / (xSigma * sqrtf(2.f));
    const float invSqrt2_y = 1.0f / (ySigma * sqrtf(2.f));
	float transmission = 0.0f;

    for (int i = 0; i < this->n_mlc_pairs; ++i) {

        const float yBottom = (this->mlc[i].y_offset - 0.5f * this->mlc[i].y_width) * mlc_scale;
        const float yTop    = (this->mlc[i].y_offset + 0.5f * this->mlc[i].y_width) * mlc_scale;

		const float xLeft  = (this->mlc[i].x1 * mlc_scale);
		const float xRight = (this->mlc[i].x2 * mlc_scale);

		const float tipMLC1 = ((xLeft - point_xyz->x) * divergence_scale) + point_xyz->x;
		const float tipMLC2 = ((xRight - point_xyz->x) * divergence_scale) + point_xyz->x;

		const float edgeMLC1 = ((yBottom - point_xyz->y) * divergence_scale) + point_xyz->y;
		const float edgeMLC2 = ((yTop - point_xyz->y) * divergence_scale) + point_xyz->y;

		const float exposedSourceX = 0.5f * (erff(tipMLC2 * invSqrt2_x) - erff(tipMLC1 * invSqrt2_x));
		const float exposedSourceY = 0.5f * (erff(edgeMLC2 * invSqrt2_y) - erff(edgeMLC1 * invSqrt2_y));

		transmission += exposedSourceX * exposedSourceY;
    }

    return transmission;

}

__host__ IMRTDose::IMRTDose(DoseClass * h_dose) : CudaDose(h_dose) {}


__global__ void termaKernel(IMRTDose * dose, IMRTBeam * beam, float * TERMAArray, float * ElectronArray){

	PointIJK vox_ijk;
	vox_ijk.k = threadIdx.x + (blockIdx.x * blockDim.x);
	vox_ijk.j = threadIdx.y + (blockIdx.y * blockDim.y);
	vox_ijk.i = threadIdx.z + (blockIdx.z * blockDim.z);

	if(!dose->pointIJKWithinImage(&vox_ijk)) {
		return;
	}

	size_t vox_index = dose->pointIJKtoIndex(&vox_ijk);

	PointXYZ vox_xyz, vox_head_xyz;
	dose->pointIJKtoXYZ(&vox_ijk, &vox_xyz, beam);
	dose->pointXYZImageToHead(&vox_xyz, &vox_head_xyz, beam);

	float distance_to_primary_source = PRIMARY_SOURCE_DISTANCE - vox_head_xyz.z;
	float distance_to_scatter_source = SCATTER_SOURCE_DISTANCE - vox_head_xyz.z;
	float primary_transmission = beam->headTransmission(&vox_head_xyz, PRIMARY_SOURCE_DISTANCE, 1.0, 1.0);
	float scatter_transmission = beam->headTransmission(&vox_head_xyz, SCATTER_SOURCE_DISTANCE, 20.0, 20.0);
	float wet = dose->WETArray[vox_index];
	float terma_primary = 0.f;
	float terma_scatter = 0.f;

	for(int i = 0; i < 12; i++){
		terma_primary += g_energy_fluence[i] * expf(-g_attenuation_coefficients[i] * wet) * powf(PRIMARY_SOURCE_DISTANCE / distance_to_primary_source, 2.0);
		terma_scatter += g_energy_fluence[i] * expf(-g_attenuation_coefficients[i] * wet * 0.2) * powf(SCATTER_SOURCE_DISTANCE / distance_to_scatter_source, 2.0);
	}

	TERMAArray[vox_index] = ((1.0 - SCATTER_SOURCE_WEIGHT) * primary_transmission * terma_primary) + (SCATTER_SOURCE_WEIGHT * scatter_transmission * terma_scatter);

	ElectronArray[vox_index] = ELECTRON_WEIGHT * (primary_transmission * (1.0 - SCATTER_SOURCE_WEIGHT) * expf(-ELECTRON_MASS_ATTENUATION * wet) + 
							SCATTER_SOURCE_WEIGHT * scatter_transmission * expf(-ELECTRON_MASS_ATTENUATION * wet));

    __syncthreads();

}

__global__ void cccKernel(IMRTDose * dose, IMRTBeam * beam, Texture3D TERMATexture, Texture3D DensityTexture, float * ElectronArray){

	PointIJK vox_ijk;
	vox_ijk.k = threadIdx.x + (blockIdx.x * blockDim.x);
	vox_ijk.j = threadIdx.y + (blockIdx.y * blockDim.y);
	vox_ijk.i = threadIdx.z + (blockIdx.z * blockDim.z);

	if(!dose->pointIJKWithinImage(&vox_ijk)) {
		return;
	}

	size_t vox_index = dose->pointIJKtoIndex(&vox_ijk);

	PointXYZ vox_img_xyz;
	dose->pointIJKtoXYZ(&vox_ijk, &vox_img_xyz, beam);

	PointXYZ tex_img_xyz;
	dose->pointXYZtoTextureXYZ(&vox_img_xyz, &tex_img_xyz, beam);

	if (TERMATexture.sample(tex_img_xyz) <= 0.01){
		dose->DoseArray[vox_index] = 0.0;
		return;
	}

	PointXYZ vox_head_xyz;
	dose->pointXYZImageToHead(&vox_img_xyz, &vox_head_xyz, beam);

	float dose_value = 0.0;
	float sp = dose->spacing / 10.0; //cm
	
	for(int i = 0; i < 6; i++){

		float th = g_kernel[0][i] * M_PI / 180.0;
		float Am = g_kernel[1][i];
		float am = g_kernel[2][i];
		float Bm = g_kernel[3][i];
		float bm = g_kernel[4][i];

		for(int j = 0; j < 12; j++){

			float phi = (float)j * 30.0 * M_PI / 180.0;
			float xr = sinf(th) * cosf(phi);
			float yr = sinf(th) * sinf(phi);
			float zr = cosf(th);

			float Rs = 0.0, Rp = 0.0, Ti = 0.0;
			float Di = AIR_DENSITY * sp;
			float ray_length = g_kernel[5][i];

			while(ray_length >= sp) {

				PointXYZ ray_head_xyz;
				ray_head_xyz.x = fmaf(xr, ray_length * 10.0, vox_head_xyz.x);
				ray_head_xyz.y = fmaf(yr, ray_length * 10.0, vox_head_xyz.y);
				ray_head_xyz.z = fmaf(zr, ray_length * 10.0, vox_head_xyz.z);

				PointXYZ ray_img_xyz;
				dose->pointXYZHeadToImage(&ray_head_xyz, &ray_img_xyz, beam);

				dose->pointXYZtoTextureXYZ(&ray_img_xyz, &tex_img_xyz, beam);
				if (dose->textureXYZWithinImage(&tex_img_xyz)) {
					Ti = TERMATexture.sample(tex_img_xyz);
					Di = DensityTexture.sample(tex_img_xyz) * sp;
				} else {
					Ti = 0.0;
					Di = AIR_DENSITY * sp;
				}

				if(Di <= 0.0){
					Di = AIR_DENSITY * sp;
				}

				Rp = Rp * exp(-am * Di) + (Ti * sinf(th) * (Am / powf(am, 2)) * (1 - exp(-am * Di)));
				Rs = Rs * (1 - (bm * Di)) + (Ti * Di * sinf(th) * (Bm / bm));
				
				ray_length = ray_length - sp;

			}

			dose_value += am * Rp + bm * Rs;

		}
	}

	dose_value += ElectronArray[vox_index];

	if(!isnan(dose_value) && (dose_value >= 0.0)){
		dose->DoseArray[vox_index] += MU_CAL * dose_value * beam->mu;
	}

	__syncthreads();

}


void photon_dose_cuda(int gpu_id, DoseClass * h_dose, BeamClass  * h_beam){

	CUDA_CHECK(hipSetDevice(gpu_id));

	IMRTDose d_dose(h_dose);
	IMRTBeam d_beam(h_beam);

	DevicePointer<float> DensityArray(h_dose->DensityArray, h_dose->num_voxels);
	DevicePointer<float> DoseArray(MemoryTag::Zeroed(), h_dose->num_voxels);
	DevicePointer<float> WETArray(MemoryTag::Zeroed(), h_dose->num_voxels);
	DevicePointer<float> TERMAArray(MemoryTag::Zeroed(), h_dose->num_voxels);
	DevicePointer<float> ElectronArray(MemoryTag::Zeroed(), h_dose->num_voxels);

	d_dose.DensityArray = DensityArray.get();
	d_dose.WETArray = WETArray.get();
	d_dose.DoseArray = DoseArray.get();
	
	DevicePointer<MLCPair> MLCPairArray(h_beam->mlc, h_beam->n_mlc_pairs);

	d_beam.mlc = MLCPairArray.get();

	DevicePointer<IMRTBeam> d_beam_ptr(&d_beam);
	DevicePointer<IMRTDose> d_dose_ptr(&d_dose);

	hipMemcpyToSymbol(HIP_SYMBOL(g_kernel), h_kernel, 6 * 6 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(g_attenuation_coefficients), h_attenuation_coefficients, 12 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(g_energy_fluence), h_energy_fluence, 12 * sizeof(float));

	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((d_dose.img_sz.k + TILE_WIDTH - 1) / TILE_WIDTH, (d_dose.img_sz.j + TILE_WIDTH - 1) / TILE_WIDTH, (d_dose.img_sz.i + TILE_WIDTH - 1) / TILE_WIDTH);

	auto DensityTexture = Texture3D::fromHostData(h_dose->DensityArray, h_dose->img_sz, hipFilterModeLinear);

    rayTraceKernel<<<dimGrid, dimBlock>>>(d_dose_ptr, d_beam_ptr, DensityTexture);
    termaKernel<<<dimGrid, dimBlock>>>(d_dose_ptr, d_beam_ptr, TERMAArray, ElectronArray);

	auto TERMATexture = Texture3D::fromDeviceData(TERMAArray, h_dose->img_sz, hipFilterModeLinear);

	cccKernel<<<dimGrid, dimBlock>>>(d_dose_ptr, d_beam_ptr, TERMATexture, DensityTexture, ElectronArray);

	CUDA_CHECK(hipMemcpy(h_dose->DoseArray, d_dose.DoseArray, d_dose.num_voxels * sizeof(float), hipMemcpyDeviceToHost));

}
